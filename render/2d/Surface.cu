#include "hip/hip_runtime.h"
#include "Surface.h"

Surface::Surface():
	data(0), pitch(0),
	surf(0)
{
	dim = make_uint2(0, 0);
}

Surface::Surface(uint2 dim)
{
	this->dim = dim;

	hipChannelFormatDesc chanel_descriptor = hipCreateChannelDesc(
		32, 32, 32, 32,
		hipChannelFormatKindFloat
	);

	hipMallocArray(
		&data, &chanel_descriptor,
		dim.x, dim.y,
		hipArraySurfaceLoadStore
	);

	pitch = dim.x * sizeof(float) * 4;

	hipResourceDesc res_desc;
	memset(&res_desc, 0, sizeof(hipResourceDesc));
	res_desc.resType = hipResourceTypeArray;
	res_desc.res.array.array = data;

	surf = 0;
	hipCreateSurfaceObject(&surf, &res_desc);
}

Surface::~Surface()
{
	if (surf != 0) hipDestroySurfaceObject(surf);
	if (data != 0) hipFreeArray(data);
}

void Surface::init(uint2 dim)
{
	if (surf != 0) return;

	this->dim = dim;

	hipChannelFormatDesc chanel_descriptor = hipCreateChannelDesc(
		32, 32, 32, 32,
		hipChannelFormatKindFloat
	);

	hipMallocArray(
		&data, &chanel_descriptor,
		dim.x, dim.y,
		hipArraySurfaceLoadStore
	);

	pitch = dim.x * sizeof(float) * 4;

	hipResourceDesc res_desc;
	memset(&res_desc, 0, sizeof(hipResourceDesc));
	res_desc.resType = hipResourceTypeArray;
	res_desc.res.array.array = data;

	surf = 0;
	hipCreateSurfaceObject(&surf, &res_desc);
}
