#include "hip/hip_runtime.h"
#include "Mask.h"
#include <string>

Mask::Mask():
	data(0), pitch(0),
	surf(0)
{
	dim = make_uint2(0, 0);
}

Mask::Mask(uint2 dim)
{
	this->dim = dim;
	auto err = hipGetLastError();
	if (err != hipSuccess) {
		std::string dd = hipGetErrorString(err);

		int a = 0;
	}
	hipChannelFormatDesc chanel_descriptor = hipCreateChannelDesc(
		32, 0, 0, 0,
		hipChannelFormatKindFloat
	);
	err = hipGetLastError();
	if (err != hipSuccess) {
		std::string dd = hipGetErrorString(err);

		int a = 0;
	}
	hipMallocArray(
		&data, &chanel_descriptor,
		dim.x, dim.y,
		hipArraySurfaceLoadStore
	);
	err = hipGetLastError();
	if (err != hipSuccess) {
		std::string dd = hipGetErrorString(err);

		int a = 0;
	}
	pitch = dim.x * sizeof(float);

	hipResourceDesc res_desc;
	memset(&res_desc, 0, sizeof(hipResourceDesc));
	res_desc.resType = hipResourceTypeArray;
	res_desc.res.array.array = data;
	err = hipGetLastError();
	if (err != hipSuccess) {
		std::string dd = hipGetErrorString(err);

		int a = 0;
	}
	surf = 0;
	hipCreateSurfaceObject(&surf, &res_desc);
	err = hipGetLastError();
	if (err != hipSuccess) {
		std::string dd = hipGetErrorString(err);

		int a = 0;
	}
}

Mask::~Mask()
{
	if (surf != 0) hipDestroySurfaceObject(surf);
	if (data != 0) hipFreeArray(data);
}

void Mask::init(uint2 dim)
{
	if (surf != 0) return;

	this->dim = dim;

	hipChannelFormatDesc chanel_descriptor = hipCreateChannelDesc(
		32, 0, 0, 0,
		hipChannelFormatKindFloat
	);

	hipMallocArray(
		&data, &chanel_descriptor,
		dim.x, dim.y,
		hipArraySurfaceLoadStore
	);

	pitch = dim.x * sizeof(float);

	hipResourceDesc res_desc;
	memset(&res_desc, 0, sizeof(hipResourceDesc));
	res_desc.resType = hipResourceTypeArray;
	res_desc.res.array.array = data;

	surf = 0;
	hipCreateSurfaceObject(&surf, &res_desc);
}

void Mask::copyFromHost(float* data)
{
	hipMemcpy2DToArray(this->data, 0, 0, data, pitch, dim.x * sizeof(float), dim.y, hipMemcpyHostToDevice);
}
