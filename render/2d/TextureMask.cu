#include "hip/hip_runtime.h"
#include "TextureMask.h"

TextureMask::TextureMask():
	texture(0), pitch(0)
{
}

TextureMask::TextureMask(uint2 dim, float* data)
{
	hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipMallocArray(&this->data, &channel_desc, dim.x, dim.y);
	pitch = dim.x * sizeof(float);

	hipMemcpy2DToArray(this->data, 0, 0, data, pitch, dim.x * sizeof(float), dim.y, hipMemcpyHostToDevice);

	struct hipResourceDesc res_desc;
	memset(&res_desc, 0, sizeof(res_desc));
	res_desc.resType = hipResourceTypeArray;
	res_desc.res.array.array = this->data;

	hipTextureDesc tex_desc;
	memset(&tex_desc, 0, sizeof(tex_desc));
	tex_desc.addressMode[0] = hipAddressModeWrap;
	tex_desc.addressMode[1] = hipAddressModeWrap;
	tex_desc.filterMode = hipFilterModeLinear;
	tex_desc.readMode = hipReadModeElementType;
	tex_desc.normalizedCoords = 1;

	hipCreateTextureObject(&texture, &res_desc, &tex_desc, NULL);

}
